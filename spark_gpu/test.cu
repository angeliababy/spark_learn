
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
__global__  void add( int a, int b, int *c )
{
    *c = a + b;
}

int main( void )
{
    int c;
    int *dev_c;
    hipMalloc( &dev_c, sizeof(int) ) ;

    add<<<1,1>>>( 2, 7, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost );
    //printf( "2 + 7 = %d\n",c);
    cout<<"2 + 7 ="<<c;
    hipFree( dev_c );

    return 0;
}
